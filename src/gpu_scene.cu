#include "cuda_api.h"
#include "cuda_error.cuh"

#include "gpu_scene.cuh"
#include "scene.h"

namespace cuwfrt
{

void GPUScene::Init(const Scene* scene)
{
    auto vectors = scene->materials.get_vectors();

    int32 offsets[Materials::count];
    int32 total_size = 0;

    for (int32 i = 0; i < vectors.size(); ++i)
    {
        offsets[i] = total_size;
        total_size += int32(vectors[i].size());
    }

    size_t offsets_size = sizeof(int32) * Materials::count;
    cudaCheck(hipMalloc(&data.offsets, offsets_size));
    cudaCheck(hipMemcpyAsync(data.offsets, offsets, offsets_size, hipMemcpyHostToDevice));

    cudaCheck(hipMalloc(&data.materials, total_size));
    for (int32 i = 0; i < Materials::count; ++i)
    {
        size_t size = vectors[i].size();
        if (size > 0)
        {
            cudaCheck(hipMemcpyAsync(data.materials + offsets[i], vectors[i].data(), size, hipMemcpyHostToDevice));
        }
    }

    size_t position_size = sizeof(Vec3) * scene->positions.size();
    cudaCheck(hipMalloc(&data.positions, position_size));
    cudaCheck(hipMemcpyAsync(data.positions, scene->positions.data(), position_size, hipMemcpyHostToDevice));

    size_t normal_size = sizeof(Vec3) * scene->normals.size();
    cudaCheck(hipMalloc(&data.normals, normal_size));
    cudaCheck(hipMemcpyAsync(data.normals, scene->normals.data(), normal_size, hipMemcpyHostToDevice));

    size_t tangent_size = sizeof(Vec3) * scene->tangents.size();
    cudaCheck(hipMalloc(&data.tangents, tangent_size));
    cudaCheck(hipMemcpyAsync(data.tangents, scene->tangents.data(), tangent_size, hipMemcpyHostToDevice));

    size_t texcoord_size = sizeof(Vec2) * scene->texcoords.size();
    cudaCheck(hipMalloc(&data.texcoords, texcoord_size));
    cudaCheck(hipMemcpyAsync(data.texcoords, scene->texcoords.data(), texcoord_size, hipMemcpyHostToDevice));

    size_t material_indices_size = sizeof(MaterialIndex) * scene->material_indices.size();
    cudaCheck(hipMalloc(&data.material_indices, material_indices_size));
    cudaCheck(
        hipMemcpyAsync(data.material_indices, scene->material_indices.data(), material_indices_size, hipMemcpyHostToDevice)
    );

    size_t indices_size = sizeof(Vec3i) * scene->indices.size();
    cudaCheck(hipMalloc(&data.indices, indices_size));
    cudaCheck(hipMemcpyAsync(data.indices, scene->indices.data(), indices_size, hipMemcpyHostToDevice));

    size_t light_indices_size = sizeof(int32) * scene->light_indices.size();
    cudaCheck(hipMalloc(&data.light_indices, light_indices_size));
    cudaCheck(hipMemcpyAsync(data.light_indices, scene->light_indices.data(), light_indices_size, hipMemcpyHostToDevice));

    // Create textures on GPU memory
    std::vector<hipTextureObject_t> temp_tex_objs;
    temp_tex_objs.reserve(scene->textures.size());
    textures.reserve(scene->textures.size());

    for (const TextureDesc& td : scene->textures)
    {
        Texture& t = textures.emplace_back(td);
        temp_tex_objs.emplace_back(t.tex_obj);
    }

    size_t textures_size = sizeof(hipTextureObject_t) * scene->textures.size();
    cudaCheck(hipMalloc(&data.tex_objs, textures_size));
    cudaCheck(hipMemcpyAsync(data.tex_objs, temp_tex_objs.data(), textures_size, hipMemcpyHostToDevice));

    // Build BVH
    BVH bvh(scene);

    size_t bvh_primitives_size = sizeof(PrimitiveIndex) * bvh.primitives.size();
    cudaCheck(hipMalloc(&data.bvh_primitives, bvh_primitives_size));
    cudaCheck(hipMemcpyAsync(data.bvh_primitives, bvh.primitives.data(), bvh_primitives_size, hipMemcpyHostToDevice));

    size_t bvh_nodes_size = sizeof(LinearBVHNode) * bvh.node_count;
    cudaCheck(hipMalloc(&data.bvh_nodes, bvh_nodes_size));
    cudaCheck(hipMemcpyAsync(data.bvh_nodes, bvh.nodes, bvh_nodes_size, hipMemcpyHostToDevice));

    cudaCheck(hipDeviceSynchronize());
}

void GPUScene::Free()
{
    cudaCheck(hipFree(data.materials));
    cudaCheck(hipFree(data.offsets));

    cudaCheck(hipFree(data.positions));
    cudaCheck(hipFree(data.normals));
    cudaCheck(hipFree(data.tangents));
    cudaCheck(hipFree(data.texcoords));
    cudaCheck(hipFree(data.material_indices));
    cudaCheck(hipFree(data.indices));
    cudaCheck(hipFree(data.light_indices));

    cudaCheck(hipFree(data.bvh_primitives));
    cudaCheck(hipFree(data.bvh_nodes));

    cudaCheck(hipFree(data.tex_objs));
}

} // namespace cuwfrt
