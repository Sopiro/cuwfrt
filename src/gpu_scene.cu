#include "api.cuh"

#include "gpu_scene.cuh"
#include "scene.h"

namespace cuwfrt
{

void GPUScene::Init(Scene* scene)
{
    size_t material_size = sizeof(Material) * scene->materials.size();
    cudaCheck(hipMalloc(&materials, material_size));
    cudaCheck(hipMemcpyAsync(materials, scene->materials.data(), material_size, hipMemcpyHostToDevice));

    size_t position_size = sizeof(Vec3) * scene->positions.size();
    cudaCheck(hipMalloc(&positions, position_size));
    cudaCheck(hipMemcpyAsync(positions, scene->positions.data(), position_size, hipMemcpyHostToDevice));

    size_t normal_size = sizeof(Vec3) * scene->normals.size();
    cudaCheck(hipMalloc(&normals, normal_size));
    cudaCheck(hipMemcpyAsync(normals, scene->normals.data(), normal_size, hipMemcpyHostToDevice));

    size_t tangent_size = sizeof(Vec3) * scene->tangents.size();
    cudaCheck(hipMalloc(&tangents, tangent_size));
    cudaCheck(hipMemcpyAsync(tangents, scene->tangents.data(), tangent_size, hipMemcpyHostToDevice));

    size_t texcoord_size = sizeof(Vec2) * scene->texcoords.size();
    cudaCheck(hipMalloc(&texcoords, texcoord_size));
    cudaCheck(hipMemcpyAsync(texcoords, scene->texcoords.data(), texcoord_size, hipMemcpyHostToDevice));

    size_t material_indices_size = sizeof(MaterialIndex) * scene->material_indices.size();
    cudaCheck(hipMalloc(&material_indices, material_indices_size));
    cudaCheck(hipMemcpyAsync(material_indices, scene->material_indices.data(), material_indices_size, hipMemcpyHostToDevice));

    size_t indices_size = sizeof(Vec3i) * scene->indices.size();
    cudaCheck(hipMalloc(&indices, indices_size));
    cudaCheck(hipMemcpyAsync(indices, scene->indices.data(), indices_size, hipMemcpyHostToDevice));

    size_t light_indices_size = sizeof(int32) * scene->light_indices.size();
    cudaCheck(hipMalloc(&light_indices, light_indices_size));
    cudaCheck(hipMemcpyAsync(light_indices, scene->light_indices.data(), light_indices_size, hipMemcpyHostToDevice));

    cudaCheck(hipDeviceSynchronize());
}

void GPUScene::Free()
{
    cudaCheck(hipFree(materials));
    cudaCheck(hipFree(positions));
    cudaCheck(hipFree(normals));
    cudaCheck(hipFree(tangents));
    cudaCheck(hipFree(texcoords));
    cudaCheck(hipFree(material_indices));
    cudaCheck(hipFree(indices));
    cudaCheck(hipFree(light_indices));
}

} // namespace cuwfrt
