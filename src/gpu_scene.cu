#include "cuda_api.h"
#include "cuda_error.cuh"

#include "gpu_scene.cuh"
#include "scene.h"

namespace cuwfrt
{

void GPUScene::Init(const Scene* scene)
{
    // Build BVH
    BVH bvh(scene);

    size_t material_size = sizeof(Material) * scene->materials.size();
    cudaCheck(hipMalloc(&materials, material_size));
    cudaCheck(hipMemcpyAsync(materials, scene->materials.data(), material_size, hipMemcpyHostToDevice));

    size_t position_size = sizeof(Vec3) * scene->positions.size();
    cudaCheck(hipMalloc(&positions, position_size));
    cudaCheck(hipMemcpyAsync(positions, scene->positions.data(), position_size, hipMemcpyHostToDevice));

    size_t normal_size = sizeof(Vec3) * scene->normals.size();
    cudaCheck(hipMalloc(&normals, normal_size));
    cudaCheck(hipMemcpyAsync(normals, scene->normals.data(), normal_size, hipMemcpyHostToDevice));

    size_t tangent_size = sizeof(Vec3) * scene->tangents.size();
    cudaCheck(hipMalloc(&tangents, tangent_size));
    cudaCheck(hipMemcpyAsync(tangents, scene->tangents.data(), tangent_size, hipMemcpyHostToDevice));

    size_t texcoord_size = sizeof(Vec2) * scene->texcoords.size();
    cudaCheck(hipMalloc(&texcoords, texcoord_size));
    cudaCheck(hipMemcpyAsync(texcoords, scene->texcoords.data(), texcoord_size, hipMemcpyHostToDevice));

    size_t material_indices_size = sizeof(MaterialIndex) * scene->material_indices.size();
    cudaCheck(hipMalloc(&material_indices, material_indices_size));
    cudaCheck(hipMemcpyAsync(material_indices, scene->material_indices.data(), material_indices_size, hipMemcpyHostToDevice));

    size_t indices_size = sizeof(Vec3i) * scene->indices.size();
    cudaCheck(hipMalloc(&indices, indices_size));
    cudaCheck(hipMemcpyAsync(indices, scene->indices.data(), indices_size, hipMemcpyHostToDevice));

    size_t light_indices_size = sizeof(int32) * scene->light_indices.size();
    cudaCheck(hipMalloc(&light_indices, light_indices_size));
    cudaCheck(hipMemcpyAsync(light_indices, scene->light_indices.data(), light_indices_size, hipMemcpyHostToDevice));

    size_t bvh_primitives_size = sizeof(PrimitiveIndex) * bvh.primitives.size();
    cudaCheck(hipMalloc(&bvh_primitives, bvh_primitives_size));
    cudaCheck(hipMemcpyAsync(bvh_primitives, bvh.primitives.data(), bvh_primitives_size, hipMemcpyHostToDevice));

    size_t bvh_nodes_size = sizeof(LinearBVHNode) * bvh.node_count;
    cudaCheck(hipMalloc(&bvh_nodes, bvh_nodes_size));
    cudaCheck(hipMemcpyAsync(bvh_nodes, bvh.nodes, bvh_nodes_size, hipMemcpyHostToDevice));

    cudaCheck(hipDeviceSynchronize());
}

void GPUScene::Free()
{
    cudaCheck(hipFree(materials));
    cudaCheck(hipFree(positions));
    cudaCheck(hipFree(normals));
    cudaCheck(hipFree(tangents));
    cudaCheck(hipFree(texcoords));
    cudaCheck(hipFree(material_indices));
    cudaCheck(hipFree(indices));
    cudaCheck(hipFree(light_indices));

    cudaCheck(hipFree(bvh_primitives));
    cudaCheck(hipFree(bvh_nodes));
}

} // namespace cuwfrt
