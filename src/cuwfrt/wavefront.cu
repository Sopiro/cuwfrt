#include "wavefront.h"

#include "cuwfrt/cuda_error.cuh"

namespace cuwfrt
{

void WavefrontResources::Init(Point2i res)
{
    ray_capacity = res.x * res.y;

    size_t ray_buffer_size = sizeof(WavefrontRay) * ray_capacity;
    size_t shadow_ray_buffer_size = sizeof(WavefrontShadowRay) * ray_capacity;

    cudaCheck(hipMalloc(&rays_active, ray_buffer_size));
    cudaCheck(hipMalloc(&rays_next, ray_buffer_size));
    cudaCheck(hipMalloc(&shadow_rays, shadow_ray_buffer_size));

    cudaCheck(hipMalloc(&active_ray_count, sizeof(int32)));
    cudaCheck(hipMalloc(&next_ray_count, sizeof(int32)));
    cudaCheck(hipMalloc(&shadow_ray_count, sizeof(int32)));
}

void WavefrontResources::Free()
{
    cudaCheck(hipFree(rays_active));
    cudaCheck(hipFree(rays_next));
    cudaCheck(hipFree(shadow_rays));

    cudaCheck(hipFree(active_ray_count));
    cudaCheck(hipFree(next_ray_count));
    cudaCheck(hipFree(shadow_ray_count));
}

void WavefrontResources::Resize(Point2i res)
{
    ray_capacity = res.x * res.y;
    cudaCheck(hipFree(rays_active));
    cudaCheck(hipFree(rays_next));
    cudaCheck(hipFree(shadow_rays));

    size_t ray_buffer_size = sizeof(WavefrontRay) * ray_capacity;
    size_t shadow_ray_buffer_size = sizeof(WavefrontShadowRay) * ray_capacity;

    cudaCheck(hipMalloc(&rays_active, ray_buffer_size));
    cudaCheck(hipMalloc(&rays_next, ray_buffer_size));
    cudaCheck(hipMalloc(&shadow_rays, shadow_ray_buffer_size));
}

} // namespace cuwfrt
