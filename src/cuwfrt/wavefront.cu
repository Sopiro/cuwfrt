#include "wavefront.h"

#include "cuwfrt/cuda_error.cuh"

namespace cuwfrt
{

void WavefrontResources::Init(Point2i res)
{
    ray_capacity = res.x * res.y;

    size_t ray_buffer_size = sizeof(WavefrontRay) * ray_capacity;
    size_t miss_ray_buffer_size = sizeof(WavefrontMissRay) * ray_capacity;
    size_t shadow_ray_buffer_size = sizeof(WavefrontShadowRay) * ray_capacity;

    cudaCheck(hipMalloc(&rays_active, ray_buffer_size));
    cudaCheck(hipMalloc(&rays_next, ray_buffer_size));

    for (size_t i = 0; i < Materials::count; ++i)
    {
        cudaCheck(hipMalloc(&rays_closest.rays[i], ray_buffer_size));
        cudaCheck(hipMalloc(&rays_closest.ray_counts[i], sizeof(int32)));
    }

    cudaCheck(hipMalloc(&miss_rays, miss_ray_buffer_size));
    cudaCheck(hipMalloc(&shadow_rays, shadow_ray_buffer_size));

    cudaCheck(hipMalloc(&active_ray_count, sizeof(int32)));
    cudaCheck(hipMalloc(&next_ray_count, sizeof(int32)));

    cudaCheck(hipMalloc(&miss_ray_count, sizeof(int32)));
    cudaCheck(hipMalloc(&shadow_ray_count, sizeof(int32)));
}

void WavefrontResources::Free()
{
    cudaCheck(hipFree(rays_active));
    cudaCheck(hipFree(rays_next));

    for (size_t i = 0; i < Materials::count; ++i)
    {
        cudaCheck(hipFree(rays_closest.rays[i]));
        cudaCheck(hipFree(rays_closest.ray_counts[i]));
    }

    cudaCheck(hipFree(miss_rays));
    cudaCheck(hipFree(shadow_rays));

    cudaCheck(hipFree(active_ray_count));
    cudaCheck(hipFree(next_ray_count));

    cudaCheck(hipFree(miss_ray_count));
    cudaCheck(hipFree(shadow_ray_count));
}

void WavefrontResources::Resize(Point2i res)
{
    ray_capacity = res.x * res.y;
    cudaCheck(hipFree(rays_active));
    cudaCheck(hipFree(rays_next));

    for (size_t i = 0; i < Materials::count; ++i)
    {
        cudaCheck(hipFree(rays_closest.rays[i]));
    }

    cudaCheck(hipFree(miss_rays));
    cudaCheck(hipFree(shadow_rays));

    size_t ray_buffer_size = sizeof(WavefrontRay) * ray_capacity;
    size_t miss_ray_buffer_size = sizeof(WavefrontMissRay) * ray_capacity;
    size_t shadow_ray_buffer_size = sizeof(WavefrontShadowRay) * ray_capacity;

    cudaCheck(hipMalloc(&rays_active, ray_buffer_size));
    cudaCheck(hipMalloc(&rays_next, ray_buffer_size));

    for (size_t i = 0; i < Materials::count; ++i)
    {
        cudaCheck(hipMalloc(&rays_closest.rays[i], ray_buffer_size));
    }

    cudaCheck(hipMalloc(&miss_rays, miss_ray_buffer_size));
    cudaCheck(hipMalloc(&shadow_rays, shadow_ray_buffer_size));
}

} // namespace cuwfrt
