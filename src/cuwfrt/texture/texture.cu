#include "hip/hip_runtime.h"
#include "texture.h"

#include "alzartak/image.h"
#include "cuwfrt/scene/gpu_scene.h"

namespace cuwfrt
{

using namespace alzartak;

Texture::Texture(const TextureDesc& td)
{
    Image4 image;
    if (td.is_constant)
    {
        image = Image4(1, 1);
        image[0] = Vec4(td.color, 1);
    }
    else
    {
        image = alzartak::ReadImage4(td.filename, td.non_color);
    }

    // Create CUDA array and texture objext
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float4>();
    hipMallocArray(&cu_array, &channel_desc, image.width, image.height);
    hipMemcpy2DToArray(
        cu_array, 0, 0, image.data.get(), image.width * sizeof(float4), image.width * sizeof(float4), image.height,
        hipMemcpyHostToDevice
    );

    // Set image resouce
    hipResourceDesc res_desc = {};
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = cu_array;

    // Set texture parameters
    hipTextureDesc tex_desc = {};
    tex_desc.addressMode[0] = hipAddressModeWrap; // wrap mode
    tex_desc.addressMode[1] = hipAddressModeWrap;
    tex_desc.filterMode = hipFilterModeLinear;    // filter mode
    tex_desc.readMode = hipReadModeElementType;
    tex_desc.normalizedCoords = 1;

    hipCreateTextureObject(&tex_obj, &res_desc, &tex_desc, nullptr);
}

Texture::~Texture()
{
    hipFreeArray(cu_array);
    hipDestroyTextureObject(tex_obj);
}

} // namespace cuwfrt
