#include "cuda_error.h"
#include "raytracer.h"

#include <hip/hip_runtime.h>

#include "cuwfrt/kernel/kernel_accumulate.cuh"
#include "cuwfrt/kernel/kernel_albedo.cuh"
#include "cuwfrt/kernel/kernel_ao.cuh"
#include "cuwfrt/kernel/kernel_debug.cuh"
#include "cuwfrt/kernel/kernel_denoise.cuh"
#include "cuwfrt/kernel/kernel_pt_naive.cuh"
#include "cuwfrt/kernel/kernel_pt_nee.cuh"
#include "cuwfrt/kernel/kernel_wavefront.cuh"

namespace cuwfrt
{

const int32 RayTracer::num_kernels = 6;
const char* RayTracer::kernel_names[] = { "Normal", "AO", "Albedo", "Pathtrace Naive", "Pathtrace NEE", "Wavefront" };

static Kernel* kernels[RayTracer::num_kernels - 1] = { RenderNormal, RaytraceAO, RaytraceAlbedo, PathTraceNaive, PathTraceNEE };

RayTracer::RayTracer(Window* window, const Scene* scene, const Camera* camera, const Options* options)
    : window{ window }
    , scene{ scene }
    , camera{ camera }
    , options{ options }
    , frame_index{ 0 }
    , spp{ 0 }
{
    res = window->GetWindowSize();

    window->SetFramebufferSizeChangeCallback([&](int32 width, int32 height) -> void { Resize(width, height); });

    for (int32 i = 0; i < 2; ++i)
    {
        frame_buffer[i].Init(res);
    }
    InitGPUResources();

    h_camera[1 - frame_index] = Camera(Point3(0), Point3(0), Vec3(0), -1, -1, -1, Point2i(-1), -1);
}

RayTracer::~RayTracer()
{
    FreeGPUResources();
    for (int32 i = 0; i < 2; ++i)
    {
        frame_buffer[i].Free();
    }
}

void RayTracer::InitGPUResources()
{
    std::cout << "Init GPU resources" << std::endl;
    gpu_res.Init(scene);
    wf.Init(res);

    const int32 capacity = res.x * res.y;
    for (int32 i = 0; i < 2; ++i)
    {
        sample_buffer[i].Init(capacity);
        g_buffer[i].Init(capacity);
        h_buffer[i].Init(capacity);
    }
    accumulation_buffer.Init(capacity);

    for (size_t i = 0; i < streams.size(); ++i)
    {
        cudaCheck(hipStreamCreate(&streams[i]));
    }

    for (size_t i = 0; i < ray_queue_streams.size(); ++i)
    {
        cudaCheck(hipStreamCreate(&ray_queue_streams[i]));
    }
}

void RayTracer::FreeGPUResources()
{
    std::cout << "Free GPU resources" << std::endl;
    gpu_res.Free();
    wf.Free();

    for (int32 i = 0; i < 2; ++i)
    {
        sample_buffer[i].Free();
        g_buffer[i].Free();
        h_buffer[i].Free();
    }
    accumulation_buffer.Free();

    for (size_t i = 0; i < streams.size(); ++i)
    {
        cudaCheck(hipStreamDestroy(streams[i]));
    }

    for (size_t i = 0; i < ray_queue_streams.size(); ++i)
    {
        cudaCheck(hipStreamDestroy(ray_queue_streams[i]));
    }
}

void RayTracer::Resize(int32 width, int32 height)
{
    if (width <= 0 || height <= 0 || (res.x == width && res.y == height))
    {
        return;
    }

    res.Set(width, height);
    glViewport(0, 0, width, height);

    cudaCheck(hipDeviceSynchronize());

    // Recreate framebuffer
    for (int32 i = 0; i < 2; ++i)
    {
        frame_buffer[i].Resize(res);
    }

    wf.Resize(res);

    const int32 capacity = res.x * res.y;
    for (int32 i = 0; i < 2; ++i)
    {
        sample_buffer[i].Resize(capacity);
        g_buffer[i].Resize(capacity);
        h_buffer[i].Resize(capacity);
    }
    accumulation_buffer.Resize(capacity);
}

void RayTracer::RayTrace(int32 kernel_index)
{
    kernel_index = Clamp(kernel_index, 0, num_kernels - 1);
    frame_index = 1 - frame_index;

    // Save camera data for motion vector calculation
    h_camera[frame_index] = *camera;

    const dim3 threads(16, 16);
    const dim3 blocks((res.x + threads.x - 1) / threads.x, (res.y + threads.y - 1) / threads.y);

    kernels[kernel_index]<<<blocks, threads>>>(
        &sample_buffer[frame_index], res, gpu_res.scene, *camera, g_buffer[frame_index], *options, spp++
    );
    cudaCheckLastError();

    cudaCheck(hipDeviceSynchronize());
}

void RayTracer::RayTraceWavefront()
{
    frame_index = 1 - frame_index;

    // Save camera data for motion vector calculation
    h_camera[frame_index] = *camera;

    int32 num_active_rays = wf.ray_capacity;
    int32 num_next_rays = 0;
    int32 num_closest_rays[wf.closest_queue_count] = { 0 };
    int32 num_miss_rays = 0;
    int32 num_shadow_rays = 0;

    // Generate Primary Rays
    {
        const dim3 threads(16, 16);
        const dim3 blocks((res.x + threads.x - 1) / threads.x, (res.y + threads.y - 1) / threads.y);
        GeneratePrimaryRays<<<blocks, threads>>>(
            wf.active.rays, &sample_buffer[frame_index], res, *camera, g_buffer[frame_index], spp++
        );
        cudaCheckLastError();
    }

    int32 bounce = 0;
    while (true)
    {
        ResetCounts<<<1, 1>>>(wf.next.count, wf.closest, wf.miss.count, wf.shadow.count);
        cudaCheckLastError();

        // Trace rays
        {
            const int32 threads = 128;
            int32 blocks = (num_active_rays + threads - 1) / threads;
            TraceRay<<<blocks, threads>>>(wf.active.rays, num_active_rays, wf.closest, wf.miss, gpu_res.scene);
            cudaCheckLastError();
        }

        // Get counts of newly generated rays (closest hit and miss)
        cudaCheck(hipMemcpy(&num_miss_rays, wf.miss.count, sizeof(int32), hipMemcpyDeviceToHost));
        for (int32 i = 0; i < wf.closest_queue_count; ++i)
        {
            cudaCheck(hipMemcpy(&num_closest_rays[i], wf.closest.counts[i], sizeof(int32), hipMemcpyDeviceToHost));
        }

        // Handle misses
        if (options->render_sky && num_miss_rays > 0)
        {
            const int32 threads = 128;
            int32 blocks = (num_miss_rays + threads - 1) / threads;
            Miss<<<blocks, threads, 0, streams[0]>>>(wf.miss.rays, num_miss_rays, &sample_buffer[frame_index]);
            cudaCheckLastError();
        }

        // Intersects closest
        for (int32 i = 0; i < wf.closest_queue_count; ++i)
        {
            if (num_closest_rays[i] > 0)
            {
                const int32 threads = 128;
                int32 blocks = (num_closest_rays[i] + threads - 1) / threads;

                DynamicDispatcher<Materials>(i).Dispatch([&](auto* m) {
                    using MaterialType = std::remove_pointer_t<decltype(m)>;
                    Closest<MaterialType><<<blocks, threads, 0, ray_queue_streams[i]>>>(
                        wf.closest.rays[i], num_closest_rays[i], wf.next, wf.shadow, &sample_buffer[frame_index], gpu_res.scene,
                        g_buffer[frame_index], bounce
                    );
                    cudaCheckLastError();
                });
            }
        }

        if (bounce++ >= options->max_bounces)
        {
            break;
        }

        for (int32 i = 0; i < wf.closest_queue_count; ++i)
        {
            cudaCheck(hipStreamSynchronize(ray_queue_streams[i]));
        }

        // Get counts of newly generated rays (shadow and next bounce)
        cudaCheck(hipMemcpyAsync(&num_shadow_rays, wf.shadow.count, sizeof(int32), hipMemcpyDeviceToHost, streams[1]));

        // Test shadow ray and incorporate direct light
        if (num_shadow_rays > 0)
        {
            const int32 threads = 128;
            int32 blocks = (num_shadow_rays + threads - 1) / threads;
            TraceShadowRay<<<blocks, threads, 0, streams[1]>>>(
                wf.shadow.rays, num_shadow_rays, &sample_buffer[frame_index], gpu_res.scene
            );
            cudaCheckLastError();
        }

        cudaCheck(hipMemcpyAsync(&num_next_rays, wf.next.count, sizeof(int32), hipMemcpyDeviceToHost));

        // Prepare for next bounce
        std::swap(wf.active, wf.next);
        num_active_rays = num_next_rays;

        if (num_active_rays <= 0)
        {
            break;
        }
    }

    cudaCheck(hipDeviceSynchronize());
}

void RayTracer::ClearSamples()
{
    spp = 0;
}

void RayTracer::AccumulateSamples(bool render)
{
    const dim3 threads(16, 16);
    const dim3 blocks((res.x + threads.x - 1) / threads.x, (res.y + threads.y - 1) / threads.y);

    Accumulate<<<blocks, threads>>>(
        &sample_buffer[frame_index], &accumulation_buffer, &frame_buffer[frame_index], res, spp, render
    );
    cudaCheckLastError();

    cudaCheck(hipDeviceSynchronize());
}

void RayTracer::Denoise()
{
    const dim3 threads(16, 16);
    const dim3 blocks((res.x + threads.x - 1) / threads.x, (res.y + threads.y - 1) / threads.y);

    int32 current_index = frame_index;
    int32 next_index = 1 - frame_index;

    PrepareDenoise<<<blocks, threads>>>(
        &accumulation_buffer, &sample_buffer[current_index], g_buffer[frame_index], h_buffer[frame_index],
        h_camera[1 - frame_index], res
    );
    cudaCheckLastError();

    static Camera camera0;
    bool consistent = (camera0 == h_camera[1 - frame_index]);

    FilterTemporal<<<blocks, threads>>>(
        &sample_buffer[current_index], res, g_buffer[1 - frame_index], g_buffer[frame_index], h_buffer[1 - frame_index],
        h_buffer[frame_index], h_camera[1 - frame_index], consistent
    );
    cudaCheckLastError();

    EstimateVariance<<<blocks, threads>>>(g_buffer[frame_index], h_buffer[frame_index], h_buffer[1 - frame_index], res);
    cudaCheckLastError();

    FilterVariance<<<blocks, threads>>>(h_buffer[1 - frame_index], h_buffer[frame_index], res);
    cudaCheckLastError();

    const int32 atrous_iterations = 5;

    for (int32 i = 0; i < atrous_iterations; ++i)
    {
        int32 step = 1 << i;

        FilterSpatial<<<blocks, threads>>>(
            &sample_buffer[current_index], &sample_buffer[next_index], res, step, g_buffer[frame_index], h_buffer[current_index],
            h_buffer[1 - current_index], spp
        );
        cudaCheckLastError();

        current_index = next_index;
        next_index = 1 - next_index;
    }

    FinalizeDenoise<<<blocks, threads>>>(&sample_buffer[current_index], res, g_buffer[frame_index]);
    cudaCheckLastError();

    TemporalAntiAliasing<<<blocks, threads>>>(
        &frame_buffer[1 - frame_index], &frame_buffer[frame_index], &sample_buffer[current_index], res, g_buffer[1 - frame_index],
        g_buffer[frame_index], consistent
    );
    cudaCheckLastError();

    // Save camera for consistency checking
    camera0 = *camera;

    cudaCheck(hipDeviceSynchronize());
}

void RayTracer::RenderAccumulated()
{
    const dim3 threads(16, 16);
    const dim3 blocks((res.x + threads.x - 1) / threads.x, (res.y + threads.y - 1) / threads.y);

    RenderFrameBuffer<<<blocks, threads>>>(&accumulation_buffer, &frame_buffer[frame_index], res);
    cudaCheckLastError();
}

void RayTracer::DrawFrame()
{
    // Copy PBO data to texture
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, frame_buffer[frame_index].pbo);
    glBindTexture(GL_TEXTURE_2D, frame_buffer[frame_index].texture);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, res.x, res.y, GL_RGBA, GL_FLOAT, nullptr);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    // OpenGL Rendering: Use PBO texture on a fullscreen quad
    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, frame_buffer[frame_index].texture);
    qr.Draw();
}

} // namespace cuwfrt
