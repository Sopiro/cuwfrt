#include "cuwfrt/cuda_api.h"
#include "cuwfrt/cuda_error.cuh"
#include "cuwfrt/util/async_job.h"

#include "gpu_scene.cuh"
#include "scene.cuh"

namespace cuwfrt
{

void GPUData::Init(const Scene* cpu_scene)
{
    // Build BVH asynchronously
    std::unique_ptr<BVH> bvh;
    auto j = RunAsync([cpu_scene, &bvh]() {
        bvh = std::make_unique<BVH>(cpu_scene);
        return true;
    });

    auto vectors = cpu_scene->materials.get_vectors();

    int32 offsets[Materials::count];
    int32 total_size = 0;

    for (int32 i = 0; i < vectors.size(); ++i)
    {
        offsets[i] = total_size;
        total_size += int32(vectors[i].size());
    }

    size_t offsets_size = sizeof(int32) * Materials::count;
    cudaCheck(hipMalloc(&scene.offsets, offsets_size));
    cudaCheck(hipMemcpyAsync(scene.offsets, offsets, offsets_size, hipMemcpyHostToDevice));

    cudaCheck(hipMalloc(&scene.materials, total_size));
    for (int32 i = 0; i < Materials::count; ++i)
    {
        size_t size = vectors[i].size();
        if (size > 0)
        {
            cudaCheck(hipMemcpyAsync(scene.materials + offsets[i], vectors[i].data(), size, hipMemcpyHostToDevice));
        }
    }

    size_t position_size = sizeof(Point3) * cpu_scene->positions.size();
    cudaCheck(hipMalloc(&scene.positions, position_size));
    cudaCheck(hipMemcpyAsync(scene.positions, cpu_scene->positions.data(), position_size, hipMemcpyHostToDevice));

    size_t normal_size = sizeof(Vec3) * cpu_scene->normals.size();
    cudaCheck(hipMalloc(&scene.normals, normal_size));
    cudaCheck(hipMemcpyAsync(scene.normals, cpu_scene->normals.data(), normal_size, hipMemcpyHostToDevice));

    size_t tangent_size = sizeof(Vec3) * cpu_scene->tangents.size();
    cudaCheck(hipMalloc(&scene.tangents, tangent_size));
    cudaCheck(hipMemcpyAsync(scene.tangents, cpu_scene->tangents.data(), tangent_size, hipMemcpyHostToDevice));

    size_t texcoord_size = sizeof(Vec2) * cpu_scene->texcoords.size();
    cudaCheck(hipMalloc(&scene.texcoords, texcoord_size));
    cudaCheck(hipMemcpyAsync(scene.texcoords, cpu_scene->texcoords.data(), texcoord_size, hipMemcpyHostToDevice));

    size_t material_indices_size = sizeof(MaterialIndex) * cpu_scene->material_indices.size();
    cudaCheck(hipMalloc(&scene.material_indices, material_indices_size));
    cudaCheck(
        hipMemcpyAsync(scene.material_indices, cpu_scene->material_indices.data(), material_indices_size, hipMemcpyHostToDevice)
    );

    size_t indices_size = sizeof(Vec3i) * cpu_scene->indices.size();
    cudaCheck(hipMalloc(&scene.indices, indices_size));
    cudaCheck(hipMemcpyAsync(scene.indices, cpu_scene->indices.data(), indices_size, hipMemcpyHostToDevice));

    size_t light_indices_size = sizeof(PrimitiveIndex) * cpu_scene->light_indices.size();
    cudaCheck(hipMalloc(&scene.light_indices, light_indices_size));
    cudaCheck(hipMemcpyAsync(scene.light_indices, cpu_scene->light_indices.data(), light_indices_size, hipMemcpyHostToDevice));

    // Create textures on GPU memory
    std::vector<hipTextureObject_t> temp_tex_objs;
    temp_tex_objs.reserve(cpu_scene->textures.size());
    textures.reserve(cpu_scene->textures.size());

    for (const TextureDesc& td : cpu_scene->textures)
    {
        Texture& t = textures.emplace_back(td);
        temp_tex_objs.emplace_back(t.tex_obj);
    }

    size_t textures_size = sizeof(hipTextureObject_t) * cpu_scene->textures.size();
    cudaCheck(hipMalloc(&scene.tex_objs, textures_size));
    cudaCheck(hipMemcpyAsync(scene.tex_objs, temp_tex_objs.data(), textures_size, hipMemcpyHostToDevice));

    j->Wait();

    size_t bvh_primitives_size = sizeof(PrimitiveIndex) * bvh->primitives.size();
    cudaCheck(hipMalloc(&scene.bvh_primitives, bvh_primitives_size));
    cudaCheck(hipMemcpyAsync(scene.bvh_primitives, bvh->primitives.data(), bvh_primitives_size, hipMemcpyHostToDevice));

    size_t bvh_nodes_size = sizeof(LinearBVHNode) * bvh->node_count;
    cudaCheck(hipMalloc(&scene.bvh_nodes, bvh_nodes_size));
    cudaCheck(hipMemcpyAsync(scene.bvh_nodes, bvh->nodes, bvh_nodes_size, hipMemcpyHostToDevice));

    cudaCheck(hipDeviceSynchronize());
}

void GPUData::Free()
{
    cudaCheck(hipFree(scene.materials));
    cudaCheck(hipFree(scene.offsets));

    cudaCheck(hipFree(scene.positions));
    cudaCheck(hipFree(scene.normals));
    cudaCheck(hipFree(scene.tangents));
    cudaCheck(hipFree(scene.texcoords));
    cudaCheck(hipFree(scene.material_indices));
    cudaCheck(hipFree(scene.indices));
    cudaCheck(hipFree(scene.light_indices));

    cudaCheck(hipFree(scene.bvh_primitives));
    cudaCheck(hipFree(scene.bvh_nodes));

    cudaCheck(hipFree(scene.tex_objs));
}

} // namespace cuwfrt
