#include "hip/hip_runtime.h"
#include "alzartak/camera.h"

#include "cuwfrt/raytracer.h"

#include "cuwfrt/scene/builder.cuh"
#include "cuwfrt/util/parallel.h"

#include "cuwfrt/loader/loader.cuh"

#include "cuwfrt/cuda_error.cuh"

using namespace alzartak;

namespace cuwfrt
{

static Window* window;
static RayTracer* raytracer;

static Scene scene;
static Camera camera;
static Options options;

static Camera3D player;

static int32 time = 0;
static int32 max_samples = 64;
static Float vfov = 71;
static Float aperture = 0;
static Float focus_dist = 1;

static int32 selection = 5;

static Vec3 GetForward()
{
    float pitch = player.rotation.x;
    float yaw = player.rotation.y;

    Vec3 forward;
    forward.x = std::cos(pitch) * std::sin(yaw);
    forward.y = -std::sin(pitch);
    forward.z = std::cos(pitch) * std::cos(yaw);

    return Normalize(forward);
}

static void Update(Float dt)
{
    window->PollEvents();

    if (player.UpdateInput(dt))
    {
        if (Length2(player.velocity) < 0.5f)
        {
            player.velocity.SetZero();
        }
        time = 0;
    }
}

static void Render()
{
    window->BeginFrame(GL_COLOR_BUFFER_BIT);

    ImGuiIO& io = ImGui::GetIO();

    // ImGui::ShowDemoWindow();

    ImGui::SetNextWindowPos({ 4, 4 }, ImGuiCond_Once, { 0.0f, 0.0f });
    if (ImGui::Begin("cuwfrt", NULL, ImGuiWindowFlags_AlwaysAutoResize))
    {
        ImGui::Text("%d fps %.6fms", int32(io.Framerate), io.DeltaTime);
        ImGui::Text("%d samples", std::min(time + 1, max_samples));
        ImGui::SetNextItemWidth(100);
        if (ImGui::SliderInt("max bounces", &options.max_bounces, 0, 64)) time = 0;
        ImGui::SetNextItemWidth(100);
        if (ImGui::SliderInt("max samples", &max_samples, 1, 1024)) time = 0;
        ImGui::Separator();
        ImGui::SetNextItemWidth(100);
        if (ImGui::SliderFloat("vfov", &vfov, 1.0f, 130.0f)) time = 0;
        ImGui::SetNextItemWidth(100);
        if (ImGui::SliderFloat("aperture", &aperture, 0.0f, 0.1f)) time = 0;
        ImGui::SetNextItemWidth(100);
        if (ImGui::SliderFloat("focus", &focus_dist, 0.0f, 10.0f)) time = 0;
        if (ImGui::Button("Reset camera", { 100, 0 }))
        {
            vfov = 71;
            aperture = 0;
            focus_dist = 1;
            time = 0;
        }
        ImGui::Separator();
        if (ImGui::Checkbox("Render sky", &options.render_sky)) time = 0;
        if (ImGui::Combo("##", &selection, RayTracer::kernel_name, RayTracer::num_kernels)) time = 0;
    }
    ImGui::End();

    camera = Camera(player.position, GetForward(), y_axis, vfov, aperture, focus_dist, window->GetWindowSize());
    if (time < max_samples)
    {
        if (selection > 4)
        {
            raytracer->RayTraceWavefront(time);
        }
        else
        {
            raytracer->RayTrace(selection, time);
        }
    }

    raytracer->DrawFrame();

    window->EndFrame();

    ++time;
}

static void BuildScene()
{
    for (int32 j = 0; j < 1; ++j)
    {
        for (int32 i = 0; i < 1; ++i)
        {
            CreateCornellBox(scene, Transform{ Point3(i * 1.1f, j * 1.1f, 0) });
        }
    }

    // static MaterialIndex mirror = scene.AddMaterial<MirrorMaterial>(Vec3{ .73f, .73f, .73f });
    // SetLoaderFallbackMaterial(mirror);

    // static MaterialIndex white = scene.AddMaterial<DiffuseMaterial>(Vec3{ .73f, .73f, .73f });
    // SetLoaderFallbackMaterial(white);

    static MaterialIndex glass = scene.AddMaterial<DielectricMaterial>(1.5f, Vec3(1.0f));
    SetLoaderFallbackMaterial(glass);

    LoadModel(
        scene, "Z:/dev/cpp_workspace/Bulbit/res/stanford/lucy.obj", Transform(Vec3(0.66f, 0.28f, -0.33f), identity, Vec3(0.5f))
    );

    // LoadModel(scene, "Z:/dev/cpp_workspace/Bulbit/res/sponza/glTF/Sponza.gltf", Transform(Vec3(0, 0, 0), identity,
    // Vec3(0.01f)));

    // LoadModel(scene, "C:/Users/sopir/Desktop/untitled.gltf", identity);
    // LoadModel(scene, "Z:/dev/cpp_workspace/Bulbit/res/stanford/bunny.obj", { Vec3(0), identity, Vec3(0.5f) });
}

static void SetImGuiStyle()
{
    // auto& io = ImGui::GetIO();
    // io.Fonts->AddFontFromFileTTF("C:/Windows/Fonts/arialbd.ttf", 16.0f, NULL, io.Fonts->GetGlyphRangesKorean());

    // Got from: https://github.com/ocornut/imgui/issues/707#issuecomment-468798935
    ImGuiStyle& style = ImGui::GetStyle();
    ImVec4* colors = style.Colors;

    /// 0 = FLAT APPEARENCE
    /// 1 = MORE "3D" LOOK
    int is3D = 0;

    colors[ImGuiCol_Text] = ImVec4(1.00f, 1.00f, 1.00f, 1.00f);
    colors[ImGuiCol_TextDisabled] = ImVec4(0.40f, 0.40f, 0.40f, 1.00f);
    colors[ImGuiCol_ChildBg] = ImVec4(0.25f, 0.25f, 0.25f, 1.00f);
    colors[ImGuiCol_WindowBg] = ImVec4(0.25f, 0.25f, 0.25f, 1.00f);
    colors[ImGuiCol_PopupBg] = ImVec4(0.25f, 0.25f, 0.25f, 1.00f);
    colors[ImGuiCol_Border] = ImVec4(0.12f, 0.12f, 0.12f, 0.71f);
    colors[ImGuiCol_BorderShadow] = ImVec4(1.00f, 1.00f, 1.00f, 0.06f);
    colors[ImGuiCol_FrameBg] = ImVec4(0.42f, 0.42f, 0.42f, 0.54f);
    colors[ImGuiCol_FrameBgHovered] = ImVec4(0.42f, 0.42f, 0.42f, 0.40f);
    colors[ImGuiCol_FrameBgActive] = ImVec4(0.56f, 0.56f, 0.56f, 0.67f);
    colors[ImGuiCol_TitleBg] = ImVec4(0.19f, 0.19f, 0.19f, 1.00f);
    colors[ImGuiCol_TitleBgActive] = ImVec4(0.22f, 0.22f, 0.22f, 1.00f);
    colors[ImGuiCol_TitleBgCollapsed] = ImVec4(0.17f, 0.17f, 0.17f, 0.90f);
    colors[ImGuiCol_MenuBarBg] = ImVec4(0.335f, 0.335f, 0.335f, 1.000f);
    colors[ImGuiCol_ScrollbarBg] = ImVec4(0.24f, 0.24f, 0.24f, 0.53f);
    colors[ImGuiCol_ScrollbarGrab] = ImVec4(0.41f, 0.41f, 0.41f, 1.00f);
    colors[ImGuiCol_ScrollbarGrabHovered] = ImVec4(0.52f, 0.52f, 0.52f, 1.00f);
    colors[ImGuiCol_ScrollbarGrabActive] = ImVec4(0.76f, 0.76f, 0.76f, 1.00f);
    colors[ImGuiCol_CheckMark] = ImVec4(0.65f, 0.65f, 0.65f, 1.00f);
    colors[ImGuiCol_SliderGrab] = ImVec4(0.52f, 0.52f, 0.52f, 1.00f);
    colors[ImGuiCol_SliderGrabActive] = ImVec4(0.64f, 0.64f, 0.64f, 1.00f);
    colors[ImGuiCol_Button] = ImVec4(0.54f, 0.54f, 0.54f, 0.35f);
    colors[ImGuiCol_ButtonHovered] = ImVec4(0.52f, 0.52f, 0.52f, 0.59f);
    colors[ImGuiCol_ButtonActive] = ImVec4(0.76f, 0.76f, 0.76f, 1.00f);
    colors[ImGuiCol_Header] = ImVec4(0.38f, 0.38f, 0.38f, 1.00f);
    colors[ImGuiCol_HeaderHovered] = ImVec4(0.47f, 0.47f, 0.47f, 1.00f);
    colors[ImGuiCol_HeaderActive] = ImVec4(0.76f, 0.76f, 0.76f, 0.77f);
    colors[ImGuiCol_Separator] = ImVec4(0.000f, 0.000f, 0.000f, 0.137f);
    colors[ImGuiCol_SeparatorHovered] = ImVec4(0.700f, 0.671f, 0.600f, 0.290f);
    colors[ImGuiCol_SeparatorActive] = ImVec4(0.702f, 0.671f, 0.600f, 0.674f);
    colors[ImGuiCol_ResizeGrip] = ImVec4(0.26f, 0.59f, 0.98f, 0.25f);
    colors[ImGuiCol_ResizeGripHovered] = ImVec4(0.26f, 0.59f, 0.98f, 0.67f);
    colors[ImGuiCol_ResizeGripActive] = ImVec4(0.26f, 0.59f, 0.98f, 0.95f);
    colors[ImGuiCol_PlotLines] = ImVec4(0.61f, 0.61f, 0.61f, 1.00f);
    colors[ImGuiCol_PlotLinesHovered] = ImVec4(1.00f, 0.43f, 0.35f, 1.00f);
    colors[ImGuiCol_PlotHistogram] = ImVec4(0.90f, 0.70f, 0.00f, 1.00f);
    colors[ImGuiCol_PlotHistogramHovered] = ImVec4(1.00f, 0.60f, 0.00f, 1.00f);
    colors[ImGuiCol_TextSelectedBg] = ImVec4(0.73f, 0.73f, 0.73f, 0.35f);
    colors[ImGuiCol_ModalWindowDimBg] = ImVec4(0.80f, 0.80f, 0.80f, 0.35f);
    colors[ImGuiCol_DragDropTarget] = ImVec4(1.00f, 1.00f, 0.00f, 0.90f);
    colors[ImGuiCol_NavHighlight] = ImVec4(0.26f, 0.59f, 0.98f, 1.00f);
    colors[ImGuiCol_NavWindowingHighlight] = ImVec4(1.00f, 1.00f, 1.00f, 0.70f);
    colors[ImGuiCol_NavWindowingDimBg] = ImVec4(0.80f, 0.80f, 0.80f, 0.20f);

    style.PopupRounding = 3;

    style.WindowPadding = ImVec2(4, 4);
    style.FramePadding = ImVec2(6, 4);
    style.ItemSpacing = ImVec2(6, 2);

    style.ScrollbarSize = 18;

    style.WindowBorderSize = 1;
    style.ChildBorderSize = 1;
    style.PopupBorderSize = 1;
    style.FrameBorderSize = float(is3D);

    style.WindowRounding = 3;
    style.ChildRounding = 3;
    style.FrameRounding = 3;
    style.ScrollbarRounding = 2;
    style.GrabRounding = 3;

#ifdef IMGUI_HAS_DOCK
    style.TabBorderSize = is3D;
    style.TabRounding = 3;

    colors[ImGuiCol_DockingEmptyBg] = ImVec4(0.38f, 0.38f, 0.38f, 1.00f);
    colors[ImGuiCol_Tab] = ImVec4(0.25f, 0.25f, 0.25f, 1.00f);
    colors[ImGuiCol_TabHovered] = ImVec4(0.40f, 0.40f, 0.40f, 1.00f);
    colors[ImGuiCol_TabActive] = ImVec4(0.33f, 0.33f, 0.33f, 1.00f);
    colors[ImGuiCol_TabUnfocused] = ImVec4(0.25f, 0.25f, 0.25f, 1.00f);
    colors[ImGuiCol_TabUnfocusedActive] = ImVec4(0.33f, 0.33f, 0.33f, 1.00f);
    colors[ImGuiCol_DockingPreview] = ImVec4(0.85f, 0.85f, 0.85f, 0.28f);

    if (ImGui::GetIO().ConfigFlags & ImGuiConfigFlags_ViewportsEnable)
    {
        style.WindowRounding = 0.0f;
        style.Colors[ImGuiCol_WindowBg].w = 1.0f;
    }
#endif
}

static void InitCudaDevice()
{
    int device_count = 0;
    cudaCheck(hipGetDeviceCount(&device_count));

    if (device_count == 0)
    {
        std::cerr << "No CUDA devices found!" << std::endl;
        exit(1);
    }

    std::cout << "Available CUDA devices: " << device_count << std::endl;

    int best_device = 0;
    int best_score = -1;

    for (int i = 0; i < device_count; ++i)
    {
        hipDeviceProp_t props;
        cudaCheck(hipGetDeviceProperties(&props, i));

        int score = 0;
        score += props.multiProcessorCount * 1000;
        score += int(props.totalGlobalMem) / (1024 * 1024 * 128);

        std::cout << i << ": " << props.name << " (SMs: " << props.multiProcessorCount
                  << ", Mem: " << (props.totalGlobalMem >> 20) << " MB)" << std::endl;

        if (score > best_score)
        {
            best_score = score;
            best_device = i;
        }
    }

    std::cout << "CUDA device set to " << best_device << std::endl;
    cudaCheck(hipSetDevice(best_device));
}

static void Init()
{
    ThreadPool::global_thread_pool.reset(new ThreadPool(std::thread::hardware_concurrency()));

    InitCudaDevice();

    window = Window::Init(1280, 720, "cuda RTRT");

    SetImGuiStyle();

    // Enable culling
    glEnable(GL_CULL_FACE);
    glCullFace(GL_BACK);
    glFrontFace(GL_CCW);

    // Enable blend
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    BuildScene();

    player.position.Set(0.5, 0.5, 1.0f);
    player.speed = 1.5f;
    player.damping = 100.0f;

    options.max_bounces = 5;

    raytracer = new RayTracer(window, &scene, &camera, &options);

    scene.Clear();
}

static void Terminate()
{
    delete raytracer;
}

} // namespace cuwfrt

int main()
{
#if defined(_WIN32) && defined(_DEBUG)
    // Enable memory-leak reports
    _CrtSetDbgFlag(_CRTDBG_ALLOC_MEM_DF | _CRTDBG_LEAK_CHECK_DF);
#endif

    using namespace cuwfrt;

    Init();

    auto last_time = std::chrono::steady_clock::now();
    const float target_frame_time = 1.0f / window->GetRefreshRate();
    float passed_time = 0;

    while (!window->ShouldClose())
    {
        auto current_time = std::chrono::steady_clock::now();
        std::chrono::duration<float> duration = current_time - last_time;
        float elapsed_time = duration.count();
        passed_time += elapsed_time;
        last_time = current_time;

        if (passed_time > target_frame_time)
        {
            while (passed_time > target_frame_time)
            {
                Update(target_frame_time);
                passed_time -= target_frame_time;
            }
            Render();
        }
    }

    Terminate();

    return 0;
}
