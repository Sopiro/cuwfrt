#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

#include "api.h"

__kernel__ void add(int n, float* x, float* y, float* z)
{
    uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        z[i] = x[i] + y[i];
    }
}

int main(void)
{
    int N = 1 << 20; // 1,048,576

    float *x, *y, *z;
    float *d_x, *d_y, *d_z;

    // Allocate memory on host (CPU)
    x = new float[N];
    y = new float[N];
    z = new float[N];

    // Allocate memory on device (GPU)
    checkCudaErrors(hipMalloc(&d_x, N * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_y, N * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_z, N * sizeof(float)));

    // Initialize x and y arrays on the host (CPU)
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
        z[i] = 0;
    }

    // Copy x and y arrays from host to device
    checkCudaErrors(hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice));

    int threads = 256; // More optimal thread count for modern GPUs
    int blocks = (N + threads - 1) / threads;

    std::cout << blocks << ", " << threads << std::endl;

    // Run kernel on 1M elements on the GPU
    add<<<blocks, threads>>>(N, d_x, d_y, d_z);

    // Check for kernel launch errors
    checkCudaErrors(hipGetLastError());

    // Wait for GPU to finish before accessing on host
    checkCudaErrors(hipDeviceSynchronize());

    // Copy result from device to host
    checkCudaErrors(hipMemcpy(z, d_z, N * sizeof(float), hipMemcpyDeviceToHost));

    // Check for errors (all values should be 3.0f)
    float max_error = 0.0f;
    for (int i = 0; i < N; i++)
    {
        max_error = std::fmax(max_error, std::fabs(z[i] - 3.0f));
    }

    std::cout << "Max error: " << max_error << std::endl;

    // Free memory on device (GPU)
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_z));

    // Free memory on host (CPU)
    delete[] x;
    delete[] y;
    delete[] z;

    return 0;
}
